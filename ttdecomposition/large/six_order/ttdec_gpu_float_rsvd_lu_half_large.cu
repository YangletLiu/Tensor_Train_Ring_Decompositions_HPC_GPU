#include "hip/hip_runtime.h"
#include "head.h"

bool reduceDim =true;
bool calError = true;
hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
hipblasGemmAlgo_t algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP; //CUBLAS_GEMM_DEFAULT_TENSOR_OP HIPBLAS_GEMM_DEFAULT
hipblasMath_t mathMode = HIPBLAS_TENSOR_OP_MATH; //HIPBLAS_TENSOR_OP_MATH  HIPBLAS_DEFAULT_MATH
hipError_t cudaStat1 = hipSuccess;
hipError_t cudaStat2 = hipSuccess;
hipError_t cudaStat3 = hipSuccess;
hipError_t cudaStat4 = hipSuccess;
hipError_t cudaStat5 = hipSuccess;
hipError_t cudaStat6 = hipSuccess;
hipError_t cudaStat7 = hipSuccess;
const float alpha = 1.0, beta0 = 0.0, beta_1 = -1.0, beta1 = 1.0;


//在TT分解中返回的是U 然后利用
void basicEig(hipsolverHandle_t cusolverH, dt *d_A, const int &m, dt *d_W){  
    int *devInfo = NULL;
    dt *d_work = NULL;
    int  lwork = 0;
    int info_gpu = 0;

    hipMalloc ((void**)&devInfo, sizeof(int));
    
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    cusolver_status = hipsolverDnSsyevd_bufferSize(
        cusolverH,
        jobz,
        uplo,
        m,
        d_A,
        m,
        d_W,
        &lwork);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cudaStat1 = hipMalloc((void**)&d_work, sizeof(dt)*lwork);
    assert(hipSuccess == cudaStat1);

    // printMatrix_Device(10, 10, d_A, 10, "A");
// step 4: compute spectrum
    cusolver_status = hipsolverDnSsyevd(
        cusolverH,
        jobz,
        uplo,
        m,
        d_A,
        m,
        d_W,
        d_work,
        lwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    // printMatrix_Device(10, 10, d_A, 10, "A2");
    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    printf("after syevd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);

    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);
    // if (d_W    ) hipFree(d_W);
}

void largeMatrixSelfMulti_once_device(hipblasHandle_t cublasH, dt *d_A,const int &m,const long long &n, dt *d_AAT){
    half *d_Ah = NULL;
    cudaStat1 = hipMalloc((void**)&d_Ah, sizeof(half)*m*n);
    assert(cudaStat1 == hipSuccess);
    f2h(d_A, d_Ah, m*n);
    cublas_status = hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T,
                           m, m, n,
                           &alpha, d_Ah, HIP_R_16F, m,
                           d_Ah, HIP_R_16F, m,
                           &beta1, d_AAT, HIP_R_32F, m,
                           HIP_R_32F, algo);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    if(d_Ah     ) hipFree(d_Ah); d_Ah=NULL;
}

void smallargeMatrixMulti_once_device(hipblasHandle_t cublasH, dt *d_A, dt *d_B, const int &ttrank, const int &m, const long long &n, dt *d_ATB){
    half *d_Ah = NULL, *d_Bh = NULL;
    cudaStat1 = hipMalloc((void**)&d_Ah, sizeof(half)*m*ttrank);
    cudaStat2 = hipMalloc((void**)&d_Bh, sizeof(half)*m*n);
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    f2h(d_A, d_Ah, m*ttrank);
    f2h(d_B, d_Bh, m*n);

    cublas_status = hipblasGemmEx(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                           ttrank, n, m,
                           &alpha, d_Ah, HIP_R_16F, m,
                           d_Bh, HIP_R_16F, m,
                           &beta0, d_ATB, HIP_R_32F, ttrank,
                           HIP_R_32F, algo);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    if(d_Ah     ) hipFree(d_Ah); d_Ah=NULL;
    if(d_Bh     ) hipFree(d_Bh); d_Bh=NULL;
}

void smallargeMatrixMulti_slice_device(hipblasHandle_t cublasH, dt *d_A, dt *d_B, const int &ttrank, const int &m, const long long &n, dt *d_ATB, const int &p){
    half *d_Ah = NULL, *d_Bh = NULL;

    long long slice = n/p;
    cudaStat1 = hipMalloc((void**)&d_Ah, sizeof(half)*m*ttrank);
    cudaStat2 = hipMalloc((void**)&d_Bh, sizeof(half)*m*n);
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    f2h(d_A, d_Ah, m*ttrank);
    f2h(d_B, d_Bh, m*n);
    //slice by slice
    for(int i = 0; i<p; i++){
        cublas_status = hipblasGemmEx(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                           ttrank, slice, m,
                           &alpha, d_Ah, HIP_R_16F, m,
                           d_Bh+i*m*slice, HIP_R_16F, m,
                           &beta0, d_ATB+slice*i*ttrank, HIP_R_32F, ttrank,
                           HIP_R_32F, algo); 
        assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    }
    if(d_Ah     ) hipFree(d_Ah); d_Ah=NULL;
    if(d_Bh     ) hipFree(d_Bh); d_Bh=NULL;
}

void longMatrixSVD_Eign_once_device(hipblasHandle_t cublasH, hipsolverHandle_t cusolverH, dt *d_A, const int &m, const long long &n, const int &ttRank, dt *d_G, dt *d_A2, const int &p){
    dt *d_W=NULL, *d_AAT=NULL;
    cudaStat1 = hipMalloc((void**)&d_AAT, sizeof(dt)*m*m);
    cudaStat2 = hipMalloc((void**)&d_W, sizeof(dt)*m);

     // printMatrix_Device( m, n, d_A, m, "A");
    largeMatrixSelfMulti_once_device(cublasH, d_A, m, n, d_AAT);
    // printMatrix_Device( m, m, d_AAT, m, "AAT");
    basicEig(cusolverH, d_AAT, m, d_W);
     // printMatrix_Device( m, m, d_AAT, m, "AAT");
    if(d_W     ) hipFree(d_W); d_W = NULL;
    matrixInvertColumn(d_AAT, d_G, m, ttRank);
    if(d_AAT    ) hipFree(d_AAT);d_AAT=NULL;
    // printMatrix_Device( 10,10, d_G1, 10, "G1");
    // 大于24需要分片
    if(m>24 && n>pow(m,4)){
        smallargeMatrixMulti_slice_device(cublasH, d_G, d_A, ttRank, m, n, d_A2, p);
        // printMatrix_Device(10, 10, d_A2, 10, "A2");
    }
    else
        smallargeMatrixMulti_once_device(cublasH, d_G, d_A, ttRank, m, n, d_A2);
    // printMatrix_Device(10, 10, d_A2, 10, "A2");
}

//TODO 实现一个超大规模矩阵A的 A*AT，存储到GPU上
void largeMatrixSelfMulti_slice_host(hipblasHandle_t cublasH, dt *h_A,const int &m,const long long &n, dt *d_AAT,const int &p){
    long long slice = 0, le = 0;
    if(n%p==0){
        slice = n/p;
    }else{
        slice = n/p;
        le = n%p;
    }
    float *d_tempA =NULL; 
    half *d_Ah = NULL;
    cudaStat1 = hipMalloc((void**)&d_tempA, sizeof(dt)*m*slice);
    cudaStat2 = hipMalloc((void**)&d_Ah, sizeof(half)*m*slice);
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    // printf("%d\n", slice);
    for(int i=0;i<p;i++){
        cudaStat3 = hipMemcpyAsync(d_tempA, h_A+m*slice*i, sizeof(dt)*m*slice, hipMemcpyHostToDevice,0);
        cudaStat4 = hipDeviceSynchronize();
        assert(cudaStat3 == hipSuccess);
        assert(cudaStat4 == hipSuccess);
        //判断是否降维
        // printMatrix_Device(m, slice, d_tempA, m, "d_tempA");
        f2h(d_tempA, d_Ah, m*slice);
        cublas_status = hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T,
                           m, m, slice,
                           &alpha, d_Ah, HIP_R_16F, m,
                           d_Ah, HIP_R_16F, m,
                           &beta1, d_AAT, HIP_R_32F, m,
                           HIP_R_32F, algo);
        assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    }
    if(d_Ah     ) hipFree(d_Ah); d_Ah=NULL;
    if(d_tempA  ) hipFree(d_tempA); d_tempA=NULL;
}


void smallargeMatrixMulti_slice_host(hipblasHandle_t cublasH, dt *d_A, dt *h_B, const int &ttrank, const int &m,  const long long &n, dt *h_ATB, const int &p){
    long long slice = 0;
    if(n%p==0)
        slice = n/p;
    else
        assert("P is not slice");
    // printf("ttrank: %d, n: %d",m,ttrank,n);
    float *d_tempB = NULL, *d_tempATB = NULL; 
    half *d_Ah = NULL, *d_Bh = NULL;
    cudaStat1 = hipMalloc((void**)&d_tempB, sizeof(dt)*m*slice);
    cudaStat2 = hipMalloc((void**)&d_tempATB, sizeof(dt)*ttrank*slice);
    cudaStat3 = hipMalloc((void**)&d_Ah, sizeof(half)*m*ttrank);
    cudaStat4 = hipMalloc((void**)&d_Bh, sizeof(half)*m*slice);
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    assert(cudaStat3 == hipSuccess);
    assert(cudaStat4 == hipSuccess);

    // printMatrix_Device(m, ttrank, d_A, m, "tempA");
    f2h(d_A, d_Ah, m*ttrank);
    // printf("%d\n", slice);
    for(int i=0;i<p;i++){
        cudaStat3 = hipMemcpyAsync(d_tempB, h_B+m*slice*i, sizeof(dt)*m*slice, hipMemcpyHostToDevice,0);
        cudaStat4 = hipDeviceSynchronize();
        assert(cudaStat3 == hipSuccess);
        assert(cudaStat4 == hipSuccess);
        // printMatrix_Device(m, slice, d_tempB, m, "tempB");
        f2h(d_tempB, d_Bh, m*slice);
        cublas_status = hipblasGemmEx(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                           ttrank, slice, m,
                           &alpha, d_Ah, HIP_R_16F, m,
                           d_Bh, HIP_R_16F, m,
                           &beta0, d_tempATB, HIP_R_32F, ttrank,
                           HIP_R_32F, algo);
        assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
        // printMatrix_Device(ttrank, slice, d_tempATB, ttrank, "tempATB");

        cudaStat3 = hipMemcpyAsync(h_ATB+ttrank*slice*i, d_tempATB, sizeof(dt)*ttrank*slice, hipMemcpyDeviceToHost, 0);
        cudaStat4 = hipDeviceSynchronize();
        assert(cudaStat3 == hipSuccess);
        assert(cudaStat4 == hipSuccess);
    }
    if(d_Ah     ) hipFree(d_Ah); d_Ah=NULL;
    if(d_Bh     ) hipFree(d_Bh); d_Bh=NULL;
    if(d_tempB  ) hipFree(d_tempB); d_tempB=NULL;
    if(d_tempATB  ) hipFree(d_tempATB); d_tempATB=NULL;
}

void longMatrixSVD_Eign_once_host(hipblasHandle_t cublasH, hipsolverHandle_t cusolverH, dt *h_A, const int &m, const long long &n, const int &ttRank, dt *h_G, dt *h_A2, const int &p){
    dt *d_W=NULL, *d_AAT=NULL,*d_G=NULL;
    cudaStat1 = hipMalloc((void**)&d_AAT, sizeof(dt)*m*m);
    cudaStat2 = hipMalloc((void**)&d_W, sizeof(dt)*m);
    cudaStat3 = hipMalloc((void**)&d_G, sizeof(dt)*m*ttRank);

    // printf("selfMtrixMul\n");
    // printMatrix_Device( m, n, d_A, m, "A");
    largeMatrixSelfMulti_slice_host(cublasH, h_A, m, n, d_AAT, p);
    // printMatrix_Device( m, m, d_AAT, m, "AAT");
    // printf("basicEig\n");
    basicEig(cusolverH, d_AAT, m, d_W);
     // printMatrix_Device( m, m, d_AAT, m, "AAT");
    if(d_W     ) hipFree(d_W); d_W = NULL;
    // printf("matrixInvertColumn\n");
    matrixInvertColumn(d_AAT, d_G, m, ttRank);
    if(d_AAT    ) hipFree(d_AAT);d_AAT=NULL;
    // printMatrix_Device( 10,10, d_G1, 10, "G1");
    // 大于24需要分片

    // printf("smallargeMatrixMulti_slice_host\n");
    smallargeMatrixMulti_slice_host(cublasH, d_G, h_A, ttRank, m, n, h_A2, p);
    // printMatrix_Device(10, 10, d_A2, 10, "A2");

    cudaStat1 = hipMemcpyAsync(h_G, d_G, sizeof(dt)*m*ttRank, hipMemcpyDeviceToHost,0);
    assert(cudaStat1 == hipSuccess);
    if(d_G) hipFree(d_G);d_G=NULL;
    // printMatrix_Device(10, 10, d_A2, 10, "A2");

}


float ttdec_half_lu_host_6(dt* h_A, int *ttRanks, const int *ttDemns, const int &calculateTimes, const int &p){
    printf("Start mul \n");
    GPUTimer timer;
    timer.start();

    for(int i=0;i<calculateTimes;i++){
        printf("***************************** %d ********************************\n", i);
        hipblasHandle_t cublasH = NULL;
        hipsolverHandle_t cusolverH = NULL;

        cusolver_status = hipsolverDnCreate(&cusolverH);
        assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

        cublas_status = hipblasCreate(&cublasH);
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

        cublas_status = hipblasSetMathMode(cublasH, mathMode);
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
        
    //1
        int m = ttDemns[0];
        int n = ttDemns[1]*ttDemns[2]*ttDemns[3]*ttDemns[4]*ttDemns[5];

        dt *h_A1=NULL, *h_G1=NULL;
        cudaStat1 = hipHostAlloc((void**)&h_G1,sizeof(dt)*m*ttRanks[1],0);
        cudaStat1 = hipHostAlloc((void**)&h_A1,sizeof(dt)*n*ttRanks[1],0);
        assert(cudaStat1 == hipSuccess);
        assert(cudaStat2 == hipSuccess);
        // printMatrix_Device(m, n, d_A, m, "A");
        longMatrixSVD_Eign_once_host(cublasH, cusolverH, h_A, m, n, ttRanks[1] , h_G1, h_A1, p);
        // printMatrix(10, 10, h_G1, 10, "G1");
        // printMatrix(10, 10, h_A2, 10, "A2");

        dt *d_A2=NULL;
        cudaStat1 = hipMalloc((void**)&d_A2, sizeof(dt)*n*ttRanks[1]);
        cudaStat2 = hipMemcpyAsync(d_A2, h_A1, sizeof(dt)*n*ttRanks[1], hipMemcpyHostToDevice,0);
        // cudaStat3 = hipDeviceSynchronize();
        assert(cudaStat1 == hipSuccess);
        assert(cudaStat2 == hipSuccess);
        if(h_A1     ) hipHostFree(h_A1);h_A1 = NULL;

    //1->2
        dt *d_G2=NULL, *d_A3=NULL, *h_G2=NULL;
        m = ttRanks[1] * ttDemns[1];
        n = n/ttDemns[1];
        cudaStat1 = hipMalloc((void**)&d_G2, sizeof(dt)*m*ttRanks[2]);
        cudaStat2 = hipHostAlloc((void**)&h_G2,sizeof(dt)*m*ttRanks[2],0);
        cudaStat2 = hipMalloc((void**)&d_A3, sizeof(dt)*n*ttRanks[2]);
        assert(cudaStat1 == hipSuccess);
        assert(cudaStat2 == hipSuccess);
        // printMatrix_Device(ttRanks[1]*m, n, d_A2, ttRanks[1]*m, "A22");
        longMatrixSVD_Eign_once_device(cublasH, cusolverH, d_A2, m, n, ttRanks[2] , d_G2, d_A3, p);
        // printMatrix_Device(ttRanks[1]*m, ttRanks[2], d_G2, ttRanks[1]*m, "G22");
        // printMatrix_Device(ttRanks[2], n, d_A3, ttRanks[2], "A32");
        if(d_A2     ) hipFree(d_A2); d_A2 = NULL;
        cudaStat5 = hipMemcpyAsync(h_G2, d_G2, sizeof(dt)*m*ttRanks[2], hipMemcpyDeviceToHost,0);
        // cudaStat6 = hipDeviceSynchronize();
        assert(cudaStat5 == hipSuccess);
        // assert(cudaStat6 == hipSuccess);
        if(d_G2     ) hipFree(d_G2); d_G2 = NULL;

    //1->2->3
        dt *d_G3=NULL, *d_A4=NULL, *h_G3=NULL;
        m = ttRanks[2] * ttDemns[2];
        n = n/ttDemns[2];
        cudaStat1 = hipMalloc((void**)&d_G3, sizeof(dt)*m*ttRanks[3]);
        cudaStat2 = hipHostAlloc((void**)&h_G3,sizeof(dt)*m*ttRanks[3],0);
        cudaStat2 = hipMalloc((void**)&d_A4, sizeof(dt)*ttRanks[3]*n);
        assert(cudaStat1 == hipSuccess);
        assert(cudaStat2 == hipSuccess);
        // printMatrix_Device(ttRanks[1]*m, n, d_A2, ttRanks[1]*m, "A22");
        longMatrixSVD_Eign_once_device(cublasH, cusolverH, d_A3, m, n, ttRanks[3] , d_G3, d_A4, p);
        // printMatrix_Device(ttRanks[1]*m, ttRanks[2], d_G2, ttRanks[1]*m, "G22");
        // printMatrix_Device(ttRanks[2], n, d_A3, ttRanks[2], "A32");
        if(d_A3     ) hipFree(d_A3); d_A3 = NULL;
        cudaStat5 = hipMemcpyAsync(h_G3, d_G3, sizeof(dt)*m*ttRanks[3], hipMemcpyDeviceToHost,0);
        // cudaStat6 = hipDeviceSynchronize();
        assert(cudaStat5 == hipSuccess);
        // assert(cudaStat6 == hipSuccess);
        if(d_G3     ) hipFree(d_G3); d_G3 = NULL;

    //1->2->3->4
        dt *d_G4=NULL, *d_A5=NULL, *h_G4=NULL;
        m = ttRanks[3] * ttDemns[3];
        n = n/ttDemns[3];
        cudaStat1 = hipMalloc((void**)&d_G4, sizeof(dt)*m*ttRanks[4]);
        cudaStat2 = hipHostAlloc((void**)&h_G4,sizeof(dt)*m*ttRanks[4],0);
        cudaStat2 = hipMalloc((void**)&d_A5, sizeof(dt)*ttRanks[4]*n);
        assert(cudaStat1 == hipSuccess);
        assert(cudaStat2 == hipSuccess);
        // printMatrix_Device(ttRanks[1]*m, n, d_A2, ttRanks[1]*m, "A22");
        longMatrixSVD_Eign_once_device(cublasH, cusolverH, d_A4, m, n, ttRanks[4] , d_G4, d_A5, p);
        // printMatrix_Device(ttRanks[1]*m, ttRanks[2], d_G2, ttRanks[1]*m, "G22");
        // printMatrix_Device(ttRanks[2], n, d_A3, ttRanks[2], "A32");
        if(d_A4     ) hipFree(d_A4); d_A4 = NULL;
        cudaStat5 = hipMemcpyAsync(h_G4, d_G4, sizeof(dt)*m*ttRanks[4], hipMemcpyDeviceToHost,0);
        // cudaStat6 = hipDeviceSynchronize();
        assert(cudaStat5 == hipSuccess);
        // assert(cudaStat6 == hipSuccess);
        if(d_G4     ) hipFree(d_G4); d_G4 = NULL;

    //1->2->3->4->5
        dt *d_G5=NULL, *d_G6=NULL, *h_G5=NULL, *h_G6=NULL;
        m = ttRanks[4] * ttDemns[4];
        n = n/ttDemns[4];
        cudaStat1 = hipMalloc((void**)&d_G5, sizeof(dt)*m*ttRanks[5]);
        cudaStat2 = hipHostAlloc((void**)&h_G5,sizeof(dt)*m*ttRanks[5],0);
        cudaStat2 = hipMalloc((void**)&d_G6, sizeof(dt)*ttRanks[5]*n);
        cudaStat2 = hipHostAlloc((void**)&h_G6,sizeof(dt)*n*ttRanks[5],0);
        assert(cudaStat1 == hipSuccess);
        assert(cudaStat2 == hipSuccess);
        // printMatrix_Device(ttRanks[1]*m, n, d_A2, ttRanks[1]*m, "A22");
        longMatrixSVD_Eign_once_device(cublasH, cusolverH, d_A5, m, n, ttRanks[5] , d_G5, d_G6, p);
        // printMatrix_Device(ttRanks[1]*m, ttRanks[2], d_G2, ttRanks[1]*m, "G22");
        // printMatrix_Device(ttRanks[5], n, d_G6, ttRanks[5], "G6");
        if(d_A5     ) hipFree(d_A5); d_A5 = NULL;
        cudaStat5 = hipMemcpyAsync(h_G5, d_G5, sizeof(dt)*m*ttRanks[5], hipMemcpyDeviceToHost,0);
        cudaStat6 = hipMemcpyAsync(h_G6, d_G6, sizeof(dt)*n*ttRanks[5], hipMemcpyDeviceToHost,0);
        // cudaStat6 = hipDeviceSynchronize();
        assert(cudaStat5 == hipSuccess);
        assert(cudaStat6 == hipSuccess);
        if(d_G5     ) hipFree(d_G5); d_G5 = NULL;
        if(d_G6     ) hipFree(d_G6); d_G6 = NULL;


        if(h_G1     ) hipHostFree(h_G1);h_G1 = NULL;
        if(h_G2     ) hipHostFree(h_G2);h_G2 = NULL;
        if(h_G3     ) hipHostFree(h_G3);h_G3 = NULL;
        if(h_G4     ) hipHostFree(h_G4);h_G4 = NULL;
        if(h_G5     ) hipHostFree(h_G5);h_G5 = NULL;
        if(h_G6     ) hipHostFree(h_G6);h_G6 = NULL;
        if(cublasH  ) hipblasDestroy(cublasH);
        if(cusolverH) hipsolverDnDestroy(cusolverH); 
    }

    float time = timer.seconds()/calculateTimes;
    return time;
}

double calMSE(hipblasHandle_t cublasH, dt *h_A, dt *h_G1, dt *h_G2, dt *h_G3, dt *h_G4, dt *h_G5, dt *h_G6, const int *ttRanks, const int *ttDemns){
    dt *d_G6=NULL, *d_G5=NULL, *d_G56=NULL;
    cudaStat1 = hipMalloc((void**)&d_G6, sizeof(dt)*ttRanks[5]*ttDemns[5]*ttRanks[6]);
    cudaStat2 = hipMalloc((void**)&d_G5, sizeof(dt)*ttRanks[4]*ttDemns[4]*ttRanks[5]);
    cudaStat3 = hipMalloc((void**)&d_G56, sizeof(dt)*ttRanks[4]*ttDemns[4]*ttDemns[5]*ttRanks[6]);
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    assert(cudaStat3 == hipSuccess);
    cudaStat1 = hipMemcpy(d_G6, h_G6, sizeof(dt)*ttRanks[5]*ttDemns[5]*ttRanks[6], hipMemcpyHostToDevice);
    cudaStat1 = hipMemcpy(d_G5, h_G5, sizeof(dt)*ttRanks[4]*ttDemns[4]*ttRanks[5], hipMemcpyHostToDevice);
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    cublas_status = hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
                           ttRanks[4]*ttDemns[4], ttDemns[5]*ttRanks[6], ttRanks[5],
                           &alpha, d_G5, ttRanks[5],
                           d_G6, ttRanks[4]*ttDemns[4],
                           &beta0, d_G56, ttRanks[4]*ttDemns[4]); 

}

float ttdec_half_lu_device_6(dt* h_A, const int *ttRanks, const int *ttDemns, const int &calculateTimes, const int &p){
    printf("Start mul \n");
    GPUTimer timer;
    timer.start();
    for(int i=0;i<calculateTimes;i++){
        printf("***************************** %d ********************************\n", i);
        hipblasHandle_t cublasH = NULL;
        hipsolverHandle_t cusolverH = NULL;

        cusolver_status = hipsolverDnCreate(&cusolverH);
        assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

        cublas_status = hipblasCreate(&cublasH);
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

        cublas_status = hipblasSetMathMode(cublasH, mathMode);
        assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
        

        int m = ttDemns[0];
        int n = ttDemns[1]*ttDemns[2]*ttDemns[3]*ttDemns[4]*ttDemns[5];
        dt *d_A=NULL, *d_G1=NULL, *d_A2=NULL, *h_G1=NULL;
        cudaStat1 = hipMalloc((void**)&d_A, sizeof(dt)*m*n);
        cudaStat2 = hipHostAlloc((void**)&h_G1,sizeof(dt)*m*ttRanks[1],0);
        cudaStat3 = hipMalloc((void**)&d_G1, sizeof(dt)*m*ttRanks[1]);
        cudaStat4 = hipMalloc((void**)&d_A2, sizeof(dt)*n*ttRanks[1]);
        cudaStat5 = hipMemcpyAsync(d_A, h_A, sizeof(dt)*m*n, hipMemcpyHostToDevice,0);
        cudaStat6 = hipDeviceSynchronize();
        assert(cudaStat1 == hipSuccess);
        assert(cudaStat2 == hipSuccess);
        assert(cudaStat3 == hipSuccess);
        assert(cudaStat4 == hipSuccess);
        assert(cudaStat5 == hipSuccess);
        assert(cudaStat6 == hipSuccess);
        // printMatrix_Device(m, n, d_A, m, "A");
        longMatrixSVD_Eign_once_device(cublasH, cusolverH, d_A, m, n, ttRanks[1] , d_G1, d_A2, p);
        // printMatrix_Device(m, ttRanks[1], d_G1, m, "G1");
        // printMatrix_Device(ttRanks[1], n, d_A2, ttRanks[1], "A2");
        if(d_A      ) hipFree(d_A); d_A = NULL;
        cudaStat5 = hipMemcpyAsync(h_G1, d_G1, sizeof(dt)*m*ttRanks[1], hipMemcpyDeviceToHost,0);
        // cudaStat6 = hipDeviceSynchronize();
        assert(cudaStat5 == hipSuccess);
        // assert(cudaStat6 == hipSuccess);
        if(d_G1     ) hipFree(d_G1); d_G1 = NULL;

    //1->2
        dt *d_G2=NULL, *d_A3=NULL, *h_G2=NULL;
        m = ttRanks[1] * ttDemns[1];
        n = n/ttDemns[1];
        cudaStat1 = hipMalloc((void**)&d_G2, sizeof(dt)*m*ttRanks[2]);
        cudaStat2 = hipHostAlloc((void**)&h_G2,sizeof(dt)*m*ttRanks[2],0);
        cudaStat2 = hipMalloc((void**)&d_A3, sizeof(dt)*n*ttRanks[2]);
        assert(cudaStat1 == hipSuccess);
        assert(cudaStat2 == hipSuccess);
        // printMatrix_Device(ttRanks[1]*m, n, d_A2, ttRanks[1]*m, "A22");
        longMatrixSVD_Eign_once_device(cublasH, cusolverH, d_A2, m, n, ttRanks[2] , d_G2, d_A3, p);
        // printMatrix_Device(ttRanks[1]*m, ttRanks[2], d_G2, ttRanks[1]*m, "G22");
        // printMatrix_Device(ttRanks[2], n, d_A3, ttRanks[2], "A32");
        if(d_A2     ) hipFree(d_A2); d_A2 = NULL;
        cudaStat5 = hipMemcpyAsync(h_G2, d_G2, sizeof(dt)*m*ttRanks[2], hipMemcpyDeviceToHost,0);
        // cudaStat6 = hipDeviceSynchronize();
        assert(cudaStat5 == hipSuccess);
        // assert(cudaStat6 == hipSuccess);
        if(d_G2     ) hipFree(d_G2); d_G2 = NULL;

    //1->2->3
        dt *d_G3=NULL, *d_A4=NULL, *h_G3=NULL;
        m = ttRanks[2] * ttDemns[2];
        n = n/ttDemns[2];
        cudaStat1 = hipMalloc((void**)&d_G3, sizeof(dt)*m*ttRanks[3]);
        cudaStat2 = hipHostAlloc((void**)&h_G3,sizeof(dt)*m*ttRanks[3],0);
        cudaStat2 = hipMalloc((void**)&d_A4, sizeof(dt)*ttRanks[3]*n);
        assert(cudaStat1 == hipSuccess);
        assert(cudaStat2 == hipSuccess);
        // printMatrix_Device(ttRanks[1]*m, n, d_A2, ttRanks[1]*m, "A22");
        longMatrixSVD_Eign_once_device(cublasH, cusolverH, d_A3, m, n, ttRanks[3] , d_G3, d_A4, p);
        // printMatrix_Device(ttRanks[1]*m, ttRanks[2], d_G2, ttRanks[1]*m, "G22");
        // printMatrix_Device(ttRanks[2], n, d_A3, ttRanks[2], "A32");
        if(d_A3     ) hipFree(d_A3); d_A3 = NULL;
        cudaStat5 = hipMemcpyAsync(h_G3, d_G3, sizeof(dt)*m*ttRanks[3], hipMemcpyDeviceToHost,0);
        // cudaStat6 = hipDeviceSynchronize();
        assert(cudaStat5 == hipSuccess);
        // assert(cudaStat6 == hipSuccess);
        if(d_G3     ) hipFree(d_G3); d_G3 = NULL;

    //1->2->3->4
        dt *d_G4=NULL, *d_A5=NULL, *h_G4=NULL;
        m = ttRanks[3] * ttDemns[3];
        n = n/ttDemns[3];
        cudaStat1 = hipMalloc((void**)&d_G4, sizeof(dt)*m*ttRanks[4]);
        cudaStat2 = hipHostAlloc((void**)&h_G4,sizeof(dt)*m*ttRanks[4],0);
        cudaStat2 = hipMalloc((void**)&d_A5, sizeof(dt)*ttRanks[4]*n);
        assert(cudaStat1 == hipSuccess);
        assert(cudaStat2 == hipSuccess);
        // printMatrix_Device(ttRanks[1]*m, n, d_A2, ttRanks[1]*m, "A22");
        longMatrixSVD_Eign_once_device(cublasH, cusolverH, d_A4, m, n, ttRanks[4] , d_G4, d_A5, p);
        // printMatrix_Device(ttRanks[1]*m, ttRanks[2], d_G2, ttRanks[1]*m, "G22");
        // printMatrix_Device(ttRanks[2], n, d_A3, ttRanks[2], "A32");
        if(d_A4     ) hipFree(d_A4); d_A4 = NULL;
        cudaStat5 = hipMemcpyAsync(h_G4, d_G4, sizeof(dt)*m*ttRanks[4], hipMemcpyDeviceToHost,0);
        // cudaStat6 = hipDeviceSynchronize();
        assert(cudaStat5 == hipSuccess);
        // assert(cudaStat6 == hipSuccess);
        if(d_G4     ) hipFree(d_G4); d_G4 = NULL;

    //1->2->3->4->5
        dt *d_G5=NULL, *d_G6=NULL, *h_G5=NULL, *h_G6=NULL;
        m = ttRanks[4] * ttDemns[4];
        n = n/ttDemns[4];
        cudaStat1 = hipMalloc((void**)&d_G5, sizeof(dt)*m*ttRanks[5]);
        cudaStat2 = hipHostAlloc((void**)&h_G5,sizeof(dt)*m*ttRanks[5],0);
        cudaStat2 = hipMalloc((void**)&d_G6, sizeof(dt)*ttRanks[5]*n);
        cudaStat2 = hipHostAlloc((void**)&h_G6,sizeof(dt)*n*ttRanks[5],0);
        assert(cudaStat1 == hipSuccess);
        assert(cudaStat2 == hipSuccess);
        // printMatrix_Device(ttRanks[1]*m, n, d_A2, ttRanks[1]*m, "A22");
        longMatrixSVD_Eign_once_device(cublasH, cusolverH, d_A5, m, n, ttRanks[5] , d_G5, d_G6, p);
        // printMatrix_Device(ttRanks[1]*m, ttRanks[2], d_G2, ttRanks[1]*m, "G22");
        // printMatrix_Device(ttRanks[5], n, d_G6, ttRanks[5], "G6");
        if(d_A5     ) hipFree(d_A5); d_A5 = NULL;
        cudaStat5 = hipMemcpyAsync(h_G5, d_G5, sizeof(dt)*m*ttRanks[5], hipMemcpyDeviceToHost,0);
        cudaStat6 = hipMemcpyAsync(h_G6, d_G6, sizeof(dt)*n*ttRanks[5], hipMemcpyDeviceToHost,0);
        // cudaStat6 = hipDeviceSynchronize();
        assert(cudaStat5 == hipSuccess);
        assert(cudaStat6 == hipSuccess);
        if(d_G5     ) hipFree(d_G5); d_G5 = NULL;
        if(d_G6     ) hipFree(d_G6); d_G6 = NULL;


        if(h_G1     ) hipHostFree(h_G1);h_G1 = NULL;
        if(h_G2     ) hipHostFree(h_G2);h_G2 = NULL;
        if(h_G3     ) hipHostFree(h_G3);h_G3 = NULL;
        if(h_G4     ) hipHostFree(h_G4);h_G4 = NULL;
        if(h_G5     ) hipHostFree(h_G5);h_G5 = NULL;
        if(h_G6     ) hipHostFree(h_G6);h_G6 = NULL;
        if(cublasH  ) hipblasDestroy(cublasH);
        if(cusolverH) hipsolverDnDestroy(cusolverH); 
    }

    float time = timer.seconds()/calculateTimes;
    return time;
}