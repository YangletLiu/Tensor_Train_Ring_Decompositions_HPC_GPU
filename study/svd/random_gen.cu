#include "hip/hip_runtime.h"
 
#include "hiprand/hiprand_kernel.h" 
#include "hiprand/hiprand.h"
#include <assert.h>
#include <stdio.h>
#include <iostream>
#include <time.h>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, 0);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, 0);
    }

    float seconds() 
    {
        hipEventRecord(stop_, 0);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return time ;
    }
    private:
    hipEvent_t start_, stop_;
};


__global__ void kernel_set_random(hiprandState *curand_states,int width,int height)//,long clock_for_rand)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;

    if(x<0 || x>width)
    {
        return;
    }
    hiprand_init(1234,x,0,&curand_states[x]);
}

__global__ void kernel_random(float *dev_random_array,int width,int height,hiprandState *curand_states)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;

    if(x<0 || x>width)
    {
        return;
    }

    for(int y=0;y<height;y++)
    {
        int pos = y*width + x;
        dev_random_array[pos] = hiprand_normal(curand_states+x);
    }
}


int main()
{
    const int array_size_width = 1000*1000;
    const int array_size_height = 55;

    //error status
    hipError_t cuda_status;

    //only chose one GPU
    cuda_status = hipSetDevice(0);
    float *dev_random_array, *h_random_array;
    hiprandState *dev_states;
     //allocate memory on the GPU
     //
    float time = 0;
    GPUTimer timer;
    timer.start();

    cuda_status = hipMalloc((void**)&dev_random_array,sizeof(float)*array_size_width*array_size_height);
    assert(cuda_status == hipSuccess);
    cuda_status = hipMalloc((void **)&dev_states,sizeof(hiprandState)*array_size_width*array_size_height);
    assert(cuda_status == hipSuccess);
    long clock_for_rand = clock();

    dim3 threads(1024,1);
    dim3 grid((array_size_width+threads.x-1)/threads.x,1);  
    
    kernel_set_random<<<grid,threads>>>(dev_states,array_size_width,array_size_height);//,clock_for_rand);
    kernel_random<<<grid,threads>>>(dev_random_array,array_size_width,array_size_height,dev_states);
    hipDeviceSynchronize();
    time = timer.seconds();
    printf("time : %f\n", time);
    

    printf("--------------------------------------------\n");
    timer.start();
    hiprandGenerator_t gen;
    float *dev_random_array2;
    cuda_status = hipMalloc((void**)&dev_random_array2,sizeof(float)*array_size_width*array_size_height);
    assert(cuda_status == hipSuccess);

    CURAND_CALL(hiprandCreateGenerator(&gen, 
                HIPRAND_RNG_PSEUDO_DEFAULT));
    
    /* Set seed */
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 
                1234ULL));

    CURAND_CALL(hiprandGenerateNormal(gen, dev_random_array2, array_size_width*array_size_height, 0, 1));

    cuda_status = hipDeviceSynchronize();
    assert(cuda_status == hipSuccess);
    time = timer.seconds();
    printf("time2 : %f\n", time);

    cuda_status = hipHostAlloc((void**)&h_random_array, sizeof(float)*array_size_width*array_size_height, 0);
    assert(cuda_status == hipSuccess);
    // for(int i = 0 ; i<array_size_width*array_size_height ; i++){
    //     h_random_array[i] = 0 ;
    // }

    cuda_status = hipMemcpyAsync(h_random_array, dev_random_array2, sizeof(float)*array_size_width*array_size_height, 
        hipMemcpyHostToDevice, 0);
    hipDeviceSynchronize();
    assert(cuda_status == hipSuccess);
    for(int i = 0 ; i<100*5 ; i++){
        std::cout<< h_random_array[i] << " ";
    }
    
    //free
    hipHostFree(h_random_array);
    hipFree(dev_random_array);
    hipFree(dev_random_array2);
    hipFree(dev_states);
    return 0;
}