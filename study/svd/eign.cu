#include "hip/hip_runtime.h"
/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include syevd_example.cpp 
     /usr/local/cuda/bin/nvcc -lcublas -lcusolver -lcurand -std=c++11 eign.cu -o eign
 *   g++ -o a.out syevd_example.o -L/usr/local/cuda/lib64 -lcudart -lcusolver
 *
 */

#include <iostream>
#include <fstream>
#include <assert.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hiprand.h>
#include <hipfft/hipfft.h>

using namespace std;
typedef float dt;
hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
hipblasGemmAlgo_t algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP; //CUBLAS_GEMM_DEFAULT_TENSOR_OP HIPBLAS_GEMM_DEFAULT
hipblasMath_t mathMode = HIPBLAS_TENSOR_OP_MATH; //HIPBLAS_TENSOR_OP_MATH  HIPBLAS_DEFAULT_MATH
hipError_t cudaStat1 = hipSuccess;
hipError_t cudaStat2 = hipSuccess;
hipError_t cudaStat3 = hipSuccess;
hipError_t cudaStat4 = hipSuccess;
hipError_t cudaStat5 = hipSuccess;
hipError_t cudaStat6 = hipSuccess;
const dt alpha = 1.0, beta = 0.0;


struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, 0);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, 0);
    }

    float seconds() 
    {
        hipEventRecord(stop_, 0);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return time * 1e-3 ;
    }
    private:
    hipEvent_t start_, stop_;
};

void genTTTensor(dt *T,long a,long b,long c,long r){
    dt *AA,*BB,*CC;    
    hipHostAlloc((void**)&AA,sizeof(dt)*a*r,0);
    hipHostAlloc((void**)&BB,sizeof(dt)*b*r,0);
    hipHostAlloc((void**)&CC,sizeof(dt)*c*r,0);
    for(long i = 0;i<a*r;i++){
        AA[i]=rand()*0.1/(RAND_MAX*0.1);
    }
    for(long i = 0;i<b*r;i++){
        BB[i]=rand()*0.1/(RAND_MAX*0.1);
    }
    for(long i = 0;i<c*r;i++){
        CC[i]=rand()*0.1/(RAND_MAX*0.1);
    }
    dt *d_T,*d_CC,*d_BB,*d_AA;
    hipMalloc((void**)&d_AA,sizeof(dt)*a*r);
    hipMalloc((void**)&d_BB,sizeof(dt)*b*r);
    hipMalloc((void**)&d_CC,sizeof(dt)*c*r);
    hipMalloc((void**)&d_T,sizeof(dt)*a*b*c);
    hipMemcpyAsync(d_BB,BB,sizeof(dt)*b*r,hipMemcpyHostToDevice,0);
    hipMemcpyAsync(d_CC,CC,sizeof(dt)*c*r,hipMemcpyHostToDevice,0);
    hipMemcpyAsync(d_AA,AA,sizeof(dt)*a*r,hipMemcpyHostToDevice,0);
    dt *d_CKRB;
    hipMalloc((void**)&d_CKRB,sizeof(dt)*c*r*b);
    hipDeviceSynchronize();

    // printMatrix_Device(a,r,d_AA,a,"AA");
    // printMatrix_Device(b,r,d_BB,b,"BB");
    // printMatrix_Device(c,r,d_CC,c,"CC");

    //X1=A*(CkrB)'  a*r  r*(bc)
    dt alpha = 1.0;
    dt beta = 0.0;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemmStridedBatched(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,
        b,c,1,
        &alpha,
        d_BB,b,b,
        d_CC,c,c,
        &beta,d_CKRB,
        b,b*c,r);
    // printMatrix_Device(b,c*r,d_CKRB,b,"CkrB");

    hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,a,b*c,r,&alpha,d_AA,a,d_CKRB,b*c,&beta,d_T,a);
    // printMatrix_Device(a,b*c,d_T,a,"T");

    hipMemcpyAsync(T,d_T,sizeof(dt)*a*b*c,hipMemcpyDeviceToHost,0);
    hipDeviceSynchronize();

    hipFree(d_AA);
    hipFree(d_BB);
    hipFree(d_CC);
    hipFree(d_CKRB);
    hipFree(d_T);
    hipHostFree(AA);
    hipHostFree(BB);
    hipHostFree(CC);
    hipblasDestroy(handle);
}

__global__ void matrixInvertColumn(dt *d_A, dt *d_Ainv, int m, int n){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    if(i<m*n){
        int row = i%m;
        int col = i/m;
        d_Ainv[row+(n-col-1)*m]=d_A[i];
        // d_Ainv[i] = d_A[];
    }
    __syncthreads();
}

void printMatrix(int m, int n, const dt*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            dt Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
}

void printMatrix_Device(int m, int n, const dt*d_A, int lda, const char* name)
{
    dt *h_A = NULL;
    hipHostAlloc((void**)&h_A , sizeof(dt)*m*n, 1);
    hipMemcpy(h_A, d_A, sizeof(dt)*m*n, hipMemcpyDeviceToHost);
    printMatrix(m, n, h_A, lda, name);
    if(h_A) hipHostFree(h_A);
}

//在TT分解中返回的是U 然后利用
void basicEig(hipsolverHandle_t cusolverH, hipblasHandle_t cublasH, dt *d_A, const int m, dt *d_W){  
    int *devInfo = NULL;
    dt *d_work = NULL;
    int  lwork = 0;
    int info_gpu = 0;

    hipMalloc ((void**)&devInfo, sizeof(int));
    
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    cusolver_status = hipsolverDnSsyevd_bufferSize(
        cusolverH,
        jobz,
        uplo,
        m,
        d_A,
        m,
        d_W,
        &lwork);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cudaStat1 = hipMalloc((void**)&d_work, sizeof(dt)*lwork);
    assert(hipSuccess == cudaStat1);

// step 4: compute spectrum
    cusolver_status = hipsolverDnSsyevd(
        cusolverH,
        jobz,
        uplo,
        m,
        d_A,
        m,
        d_W,
        d_work,
        lwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    printf("after syevd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);

    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);
    // if (d_W    ) hipFree(d_W);
}

int main(int argc, char*argv[])
{
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    const int m = 160;
    const int n = 1600*1600;
    // const int r = 50;
    // int r_mul8 = 56;  //Multiple of 8
    // if(m > 500){
    //     r_mul8 = q/10/8*8;
    // }
    // genTTTensor(A,m,m,m,r);
    dt *A=NULL;
    hipHostMalloc((void**)&A, sizeof(dt)*m*n);
    for(int i = 0; i < n*m; i++){
        A[i] = (dt) (rand()*1.0 / RAND_MAX*1.0)*2.0-1.0;
    }
    // dt V[n*m]; // eigenvectors
    // dt W[m]; // eigenvalues

    dt *d_U = NULL;
    dt *d_A = NULL;
    dt *d_W = NULL;
    dt *d_A2 = NULL;
    dt *d_Ueig = NULL;
    
// step 1: create cusolver/cublas handle
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    cublas_status = hipblasSetMathMode(cublasH, mathMode);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

// step 2: copy A and B to device
    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(dt) * m * n);
    cudaStat1 = hipMalloc ((void**)&d_Ueig, sizeof(dt) * m * m);
    cudaStat1 = hipMalloc ((void**)&d_U, sizeof(dt) * m * m);
    cudaStat1 = hipMalloc ((void**)&d_A2, sizeof(dt) * m * n);
    cudaStat2 = hipMalloc ((void**)&d_W, sizeof(dt) * m);
    
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(dt) * m * n, hipMemcpyHostToDevice);
    // printMatrix_Device(m, n, d_A, m, "A");
    assert(hipSuccess == cudaStat1);

    GPUTimer timer0;
    timer0.start();
    cublas_status = hipblasSgemm(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, m, m, n, &alpha, d_A, m, d_A, m, &beta, d_Ueig, m);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);
    printMatrix_Device(m, m, d_Ueig, m, "AAT");

    basicEig(cusolverH, cublasH, d_Ueig, m, d_W);
    if (d_W) hipFree(d_W);
    // printMatrix_Device(m, m, d_Ueig, m, "Ueig");
    dim3 threads(512,1,1);
    dim3 blocksUinv((m*m+512-1)/512,1,1);
    matrixInvertColumn<<<blocksUinv, threads>>>(d_Ueig, d_U, m, m);
    if(d_Ueig ) hipFree(d_Ueig);
    // printMatrix_Device(m, m, d_U, m, "U");
    // printf("=====\n");
 
    cublas_status = hipblasGemmEx(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N,
                           m, n, m,
                           &alpha, d_U, HIP_R_32F, m,
                           d_A, HIP_R_32F, m,
                           &beta, d_A2, HIP_R_32F, m,
                           HIP_R_32F, algo);
    assert(cublas_status == HIPBLAS_STATUS_SUCCESS);

    float time2 = timer0.seconds();
    printf("\n***********************\n time1: %f \n*********************\n", time2);

    // printMatrix_Device(m, n, d_A2, m, "SVT");
    // printf("=====\n");
    
    if (d_A    ) hipFree(d_A);
    if (d_A2  ) hipFree(d_A2);
    if (d_U    ) hipFree(d_U);
    if (A) hipHostFree(A);

    if (cusolverH) hipsolverDnDestroy(cusolverH);
    if (cublasH ) hipblasDestroy(cublasH);
    hipDeviceReset();

    return 0;
}