#include <iostream>
#include <fstream>
#include <assert.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hiprand/hiprand.h>
#include <hipfft/hipfft.h>

using namespace std;
typedef float dt;

struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, 0);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, 0);
    }

    float seconds() 
    {
        hipEventRecord(stop_, 0);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return time * 1e-3 ;
    }
    private:
    hipEvent_t start_, stop_;
};

// nvcc -lcublas -lcusolver -lcurand -std=c++11 svd.cu -o svd
void printMatrix(int m, int n, const dt*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            float Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
    cout << " ------------------------------------" << endl;
}
// nvcc -lcublas -lcusolver -lcurand -std=c++11 svd.cu -o svd
void printMatrix_Device(int m, int n, const dt*d_A, int lda, const char* name)
{
    dt *h_A = NULL;
    hipHostAlloc((void**)&h_A , sizeof(dt)*m*n, 1);
    hipMemcpy(h_A, d_A, sizeof(dt)*m*n, hipMemcpyDeviceToHost);
    printMatrix(m, n, h_A, lda, name);
    if(h_A) hipHostFree(h_A);
}

int main(int argc, char*argv[])
{
	//for(int i=100;i<=1000;i=i+100){
    hipsolverHandle_t cusolverH = NULL;
	hipsolverDnCreate(&cusolverH);
    int m = 10000;//
    int n = 100;//  m必须>=n
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    //cout<<"size:"<<i<<endl;
    srand(time(NULL));
    float *A = NULL;  //  b*a
    hipHostAlloc((void**)&A,sizeof(float)*m*n,0);
    
    for(long i = 0; i < m*n; ++i) {
    	A[i]= (dt) (rand()*1.0 / RAND_MAX *1.0);
    }
    // float A[6] = { 1.0, 4.0, 2.0, 2.0, 5.0, 1.0};
    float *d_A;
    hipMalloc((void**)&d_A,sizeof(float)*m*n);
    cudaStat1 = hipMemcpyAsync(d_A, A,sizeof(float)*m*n,hipMemcpyHostToDevice,0);
    assert(cudaStat1 == hipSuccess);
    hipDeviceSynchronize();

    float *d_S = NULL;
    float *d_U = NULL;
    float *d_VT = NULL;
    int *devInfo = NULL;
    float *d_work = NULL;
    float *d_rwork = NULL;
    //float *d_W = NULL; 
    int lwork = 0;
    cudaStat1 = hipMalloc ((void**)&d_S  , sizeof(double)*n);
    cudaStat2 = hipMalloc ((void**)&d_U  , sizeof(double)*m*n);
    cudaStat3 = hipMalloc ((void**)&d_VT , sizeof(double)*n*n);
    cudaStat4 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    assert(cudaStat3 == hipSuccess);
    assert(cudaStat4 == hipSuccess);

    cout<< " m:" << m << " n:" << n  <<endl;
    cusolver_status=  hipsolverDnDgesvd_bufferSize(
        cusolverH,
        m,
        n,
        &lwork );
    if(HIPSOLVER_STATUS_INVALID_VALUE == cusolver_status)
        cout << "CUSOLVER_STATUS_INVALID_VALUE" <<endl;
    if(HIPSOLVER_STATUS_ARCH_MISMATCH == cusolver_status)
        cout << "CUSOLVER_STATUS_ARCH_MISMATCH" <<endl;
    if(HIPSOLVER_STATUS_INTERNAL_ERROR == cusolver_status)
        cout << "CUSOLVER_STATUS_INTERNAL_ERROR" <<endl;
    if(HIPSOLVER_STATUS_NOT_INITIALIZED == cusolver_status)
        cout << "CUSOLVER_STATUS_NOT_INITIALIZED" <<endl;
    assert(cusolver_status == HIPSOLVER_STATUS_SUCCESS);
    cudaStat1 = hipMalloc((void**)&d_work , sizeof(double)*lwork);
    assert(cudaStat1 == hipSuccess);
    signed char jobu = 'S'; 
    signed char jobvt = 'S';

    dt time = 0.0f;
    GPUTimer timer;
    timer.start();
    cusolver_status = hipsolverDnSgesvd (
        cusolverH,
        jobu,
        jobvt,
        m,
        n,
        d_A,
        m,
        d_S,
        d_U,
        m,  // ldu
        d_VT,
        n, // ldvt,
        d_work,
        lwork,
        d_rwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    if(HIPSOLVER_STATUS_INVALID_VALUE == cusolver_status)
        cout << "CUSOLVER_STATUS_INVALID_VALUE" <<endl;
    if(HIPSOLVER_STATUS_ARCH_MISMATCH == cusolver_status)
        cout << "CUSOLVER_STATUS_ARCH_MISMATCH" <<endl;
    if(HIPSOLVER_STATUS_INTERNAL_ERROR == cusolver_status)
        cout << "CUSOLVER_STATUS_INTERNAL_ERROR" <<endl;
    if(HIPSOLVER_STATUS_NOT_INITIALIZED == cusolver_status)
        cout << "CUSOLVER_STATUS_NOT_INITIALIZED" <<endl;
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);
    time = timer.seconds();
    cout << "----------------------------------\npart one svd run time: " << time << "s"<< endl;
    printMatrix_Device(n,1,d_S,n,"S");

    hipFree(d_S);
    hipFree(d_U);
    hipFree(d_VT);
    hipFree(devInfo);
    hipFree(d_work);
    hipFree(d_A);
    hipHostFree(A);
   	//cudaFree(d_rwork);
	hipsolverDnDestroy(cusolverH);
	hipDeviceSynchronize();

}
//}