#include <iostream>
#include <fstream>
#include <assert.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hiprand.h>
#include <hipfft/hipfft.h>

using namespace std;
typedef float dt;
hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
hipError_t cudaStat1 = hipSuccess;
hipError_t cudaStat2 = hipSuccess;
hipError_t cudaStat3 = hipSuccess;
hipError_t cudaStat4 = hipSuccess;
hipError_t cudaStat5 = hipSuccess;
hipError_t cudaStat6 = hipSuccess;
const float alpha = 1.0, beta = 0.0;

struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, 0);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, 0);
    }

    float seconds() 
    {
        hipEventRecord(stop_, 0);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return time * 1e-3 ;
    }
    private:
    hipEvent_t start_, stop_;
};

// /usr/local/cuda/bin/nvcc -lcublas -lcusolver -lcurand -std=c++11 svd2.cu -o svd2
void printMatrix(int m, int n, const dt*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            float Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
    cout << " ------------------------------------" << endl;
}
// nvcc -lcublas -lcusolver -lcurand -std=c++11 svd.cu -o svd
void printMatrix_Device(int m, int n, const dt*d_A, int lda, const char* name)
{
    dt *h_A = NULL;
    hipHostAlloc((void**)&h_A , sizeof(dt)*m*n, 1);
    hipMemcpy(h_A, d_A, sizeof(dt)*m*n, hipMemcpyDeviceToHost);
    printMatrix(m, n, h_A, lda, name);
    if(h_A) hipHostFree(h_A);
}


void basicSvd(hipsolverHandle_t cusolverH, hipblasHandle_t cublasH, float *d_B, const int m, const int n, float *d_UT, float *d_S, float *d_V){
    printf("start svd for m: %d, n:%d\n", m, n);
    float *d_BT = NULL, *d_U = NULL;
    float *d_work = NULL, *d_rwork = NULL;
    int *devInfo = NULL;
    int lwork = 0,  info_gpu = 0;

    cudaStat1 = hipMalloc((void**)&d_BT, sizeof(float)*m*n);
    cudaStat2 = hipMalloc((void**)&d_U, sizeof(float)*m*m);
    cudaStat3 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    assert(cudaStat3 == hipSuccess);

//转置B
    cublas_status = hipblasSgeam(cublasH,
                            HIPBLAS_OP_T, HIPBLAS_OP_N, 
                            n, m,
                            &alpha,
                            d_B, m,
                            &beta,
                            d_B, n,
                            d_BT, n);
    assert(cublas_status == HIPSOLVER_STATUS_SUCCESS);
    hipDeviceSynchronize();
    cout <<"m: "<< m <<"  n: "<< n <<endl;
    cusolver_status = hipsolverDnDgesvd_bufferSize(
        cusolverH,
        n,
        m,
        &lwork);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cudaStat1 = hipMalloc((void**)&d_work , sizeof(float)*lwork);
    cout << "svd sapce: " << lwork << "GB"<<endl;
    assert(hipSuccess == cudaStat1);

    signed char jobu = 'S'; // all m columns of U
    signed char jobvt = 'S'; // all n columns of VT
    cout << "n: " << n <<" m:" << m <<endl;
    cusolver_status = hipsolverDnSgesvd(
        cusolverH,
        jobu,
        jobvt,
        n,
        m,
        d_BT,
        n,
        d_S,
        d_V,
        n,  // ldu
        d_U,
        m, // ldvt,
        d_work,
        lwork,
        d_rwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    if(HIPSOLVER_STATUS_INVALID_VALUE == cusolver_status)
        cout << "HIPSOLVER_STATUS_INVALID_VALUE" <<endl;
    if(HIPSOLVER_STATUS_ARCH_MISMATCH == cusolver_status)
        cout << "HIPSOLVER_STATUS_ARCH_MISMATCH" <<endl;
    if(HIPSOLVER_STATUS_INTERNAL_ERROR == cusolver_status)
        cout << "HIPSOLVER_STATUS_INTERNAL_ERROR" <<endl;
    if(HIPSOLVER_STATUS_NOT_INITIALIZED == cusolver_status)
        cout << "HIPSOLVER_STATUS_NOT_INITIALIZED" <<endl;
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    // printf("after gesvd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);

//转置U 给UT
    cublas_status = hipblasSgeam(cublasH,
                            HIPBLAS_OP_T, HIPBLAS_OP_N, 
                            m, m,
                            &alpha,
                            d_U, m,
                            &beta,
                            d_U, m,
                            d_UT, m);
    assert(cublas_status == HIPSOLVER_STATUS_SUCCESS);
    hipDeviceSynchronize();

    if(d_BT) hipFree(d_BT); d_BT = NULL;
    if(d_U) hipFree(d_U); d_U = NULL;
    if(d_work) hipFree(d_work); d_work = NULL;
    if(devInfo) hipFree(devInfo); devInfo = NULL;
    if(d_rwork) hipFree(d_rwork); d_rwork = NULL;
}




int main(int argc, char*argv[])
{
	hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    
    int k = 2360;
    const int m = k/10/8*8+8;
    const long int n = k*k;
    
/*       | 1 2  |
 *   A = | 4 5  |
 *       | 2 1  |
 */
    float *A = NULL;
    srand(time(NULL));
    //float A[lda*n] = { 0.4, 0.2, 0.1, 0.5, 0.3, 0.2};
    hipHostAlloc((void**)&A, sizeof(float)*m*n,0);
    for(int i = 0; i < m*n; ++i){
        A[i] = (float) rand()*1.0 / RAND_MAX*1.0;
    }
    // float U[lda*m]; // m-by-m unitary matrix 
    // float VT[lda*n];  // n-by-n unitary matrix
    // float S[n]; // singular value
    // float S_exact[n] = {7.065283497082729, 1.040081297712078};

    float *d_A = NULL;
    float *d_S = NULL;
    float *d_U = NULL;
    float *d_VT = NULL;

    // printf("A = (matlab base-1)\n");
    // printMatrix(m, n, A, lda, "A");
    // printf("=====\n");

// step 1: create cusolverDn/cublas handle
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

// step 2: copy A and B to device
    cudaStat1 = hipMalloc ((void**)&d_A  , sizeof(float)*m*n);
    cudaStat2 = hipMalloc ((void**)&d_S  , sizeof(float)*m);
    cudaStat3 = hipMalloc ((void**)&d_U  , sizeof(float)*m*m);
    cudaStat4 = hipMalloc ((void**)&d_VT , sizeof(float)*m*n); 
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(float)*m*n, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);

    basicSvd(cusolverH, cublasH, d_A, m, n, d_U, d_S, d_VT);

// free resources
    if (d_A    ) hipFree(d_A);
    if (d_S    ) hipFree(d_S);
    if (d_U    ) hipFree(d_U);
    if (d_VT   ) hipFree(d_VT);

    if (cublasH ) hipblasDestroy(cublasH);
    if (cusolverH) hipsolverDnDestroy(cusolverH);

    hipDeviceReset();

    return 0;
}
//}