#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <hipblas.h>
#include <hipsolver.h>

using namespace std;
hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;    
hipError_t cudaStat1 = hipSuccess;
hipError_t cudaStat2 = hipSuccess;
hipError_t cudaStat3 = hipSuccess;
hipError_t cudaStat4 = hipSuccess;
typedef float dt;

struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, 0);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, 0);
    }

    float seconds() 
    {
        hipEventRecord(stop_, 0);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return time * 1e-3 ;
    }
    private:
    hipEvent_t start_, stop_;
};


// /usr/local/cuda/bin/nvcc -lcublas -lcusolver -lcurand -std=c++11 qr.cu -o qr
void printMatrix(int m, int n, const dt*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            float Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
    cout << " ------------------------------------" << endl;
}

void printMatrix_Device(int m, int n, const dt*d_A, int lda, const char* name)
{
    dt *h_A = NULL;
    hipHostAlloc((void**)&h_A , sizeof(dt)*m*n, 1);
    hipMemcpy(h_A, d_A, sizeof(dt)*m*n, hipMemcpyDeviceToHost);
    printMatrix(m, n, h_A, lda, name);
    if(h_A) hipHostFree(h_A);
}



void basicQR(hipsolverHandle_t cusolverH, float *d_A, int m, int n){
    printf("start qr for m: %d, n:%d\n", m,n);
    float *d_work = NULL, *d_tau = NULL;
    int *devInfo = NULL;
    int  lwork = 0; 
    int info_gpu = 0;
    cudaStat1 = hipMalloc((void**)&d_tau, sizeof(float)*n);
    cudaStat2 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    cusolver_status = hipsolverDnSgeqrf_bufferSize(
        cusolverH, 
        m, 
        n, 
        d_A, 
        m, 
        &lwork);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
    cout << "qr space"<< sizeof(float)*lwork/1024/1024/1024<<"GB"<<endl;
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(float)*lwork);
    assert(hipSuccess == cudaStat1);
    // assert(cudaSuccess == cudaStat2);

    cusolver_status = hipsolverDnSgeqrf(
        cusolverH, 
        m, 
        n, 
        d_A, 
        m, 
        d_tau, 
        d_work, 
        lwork, 
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    // check if QR is good or not
    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    // printf("after geqrf: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);

    cusolver_status= hipsolverDnSorgqr(
        cusolverH,
        m,
        n,
        n,
        d_A,
        m,
        d_tau,
        d_work,
        lwork,
        devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    // printf("after orgqr: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);

    if (d_work) hipFree(d_work); d_work = NULL;
    if (devInfo) hipFree(devInfo); devInfo = NULL;
    if (d_tau) hipFree(d_tau); d_tau = NULL;
}


// __global__ getMatrixUTriangular(int n, float* d_A, float* d_R){

// }

int main(int argc, char*argv[])
{
    hipsolverHandle_t cusolverH = NULL;
   
    const int m = 2360*2360;
    const int n = 240;
/*       | 1 2 3 |
 *   A = | 4 5 6 |
 *       | 2 1 1 |
 *
 *   x = (1 1 1)'
 *   b = (6 15 4)'
 */

    float *A = NULL;
    srand(time(NULL));
    //float A[lda*n] = { 0.4, 0.2, 0.1, 0.5, 0.3, 0.2};
    hipHostAlloc((void**)&A, sizeof(float)*m*n,0);
    for(int i = 0; i < m*n; ++i){
        A[i] = (float) rand()*1.0 / RAND_MAX*1.0;
    }
 
// step 1: create cusolver/cublas handle
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    
    float *d_A = NULL; // linear memory of GPU  
// step 2: copy A and B to device
    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(float) * m * n);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(d_A, A, sizeof(float) * m * n, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);

    basicQR(cusolverH, d_A, m, n);

// free resources
    if (d_A    ) hipFree(d_A);
    if (cusolverH) hipsolverDnDestroy(cusolverH);   

    hipDeviceReset();

    return 0;
}