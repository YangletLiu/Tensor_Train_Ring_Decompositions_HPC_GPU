#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <assert.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hiprand.h>
#include <hipfft/hipfft.h>
#include <hip/hip_fp16.h>

using namespace std;
typedef float dt;
bool reduceDim =true;
hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
hipblasGemmAlgo_t algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP; //CUBLAS_GEMM_DEFAULT_TENSOR_OP HIPBLAS_GEMM_DEFAULT
hipblasMath_t mathMode = HIPBLAS_TENSOR_OP_MATH; //HIPBLAS_TENSOR_OP_MATH  HIPBLAS_DEFAULT_MATH
hipError_t cudaStat1 = hipSuccess;
hipError_t cudaStat2 = hipSuccess;
hipError_t cudaStat3 = hipSuccess;
hipError_t cudaStat4 = hipSuccess;
hipError_t cudaStat5 = hipSuccess;
hipError_t cudaStat6 = hipSuccess;
const float alpha = 1.0, beta = 0.0, beta2 = -1.0;

__global__ void warmup()
{
    int sum=0;
    for(int i = 0; i < 1000; i++) {
        sum+=i;
    }
}   

void warmupcu(){
    warmup<<<1,1>>>();
}

void norm2HH(float *A, long long len, float *norm2){
  // printf("%lld\n", len);
  double norm2Result = 0.0;
  for(long long i = 0; i < len; ++i){
    norm2Result += (A[i] * A[i]);
  }
  *norm2 = sqrtf(norm2Result);
}

void norm2HH_device(float *d_A, long long len, float *norm2){
    // printf("%lld\n", len);
    dt *h_A = NULL;
    hipHostAlloc((void**)&h_A , sizeof(dt)*len, 1);
    hipMemcpy(h_A, d_A, sizeof(dt)*len, hipMemcpyDeviceToHost);
    norm2HH(h_A, len, norm2);
    if(h_A) hipHostFree(h_A);
}


struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, 0);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, 0);
    }

    float seconds()
    {
        hipEventRecord(stop_, 0);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return time*1e-3;
    }
    private:
    hipEvent_t start_, stop_;
};

// /usr/local/cuda/bin/nvcc -lcublas -lcusolver -lcurand -std=c++11 largeMatrixMulti_AAT.cu -o largeMatrixMulti_AAT
void printMatrix(int m, int n, const dt*A, int lda, const char* name)
{
    for(int row = 0 ; row < m ; row++){
        for(int col = 0 ; col < n ; col++){
            float Areg = A[row + col*lda];
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg);
        }
    }
    cout << " ------------------------------------" << endl;
}
// nvcc -lcublas -lcusolver -lcurand -std=c++11 svd.cu -o svd
void printMatrix_Device(int m, int n, const dt*d_A, int lda, const char* name)
{
    dt *h_A = NULL;
    hipHostAlloc((void**)&h_A , sizeof(dt)*m*n, 1);
    hipMemcpy(h_A, d_A, sizeof(dt)*m*n, hipMemcpyDeviceToHost);
    printMatrix(m, n, h_A, lda, name);
    if(h_A) hipHostFree(h_A);
}

__global__  void floattohalf(dt *AA,half *BB,long m){
    long i = blockIdx.x*blockDim.x+threadIdx.x;
    const long temp = blockDim.x*gridDim.x;
    if(i<m){
        BB[i] = __float2half(AA[i]);
        i+=temp;
    }
    __syncthreads();
}

void f2h(dt *A,half *B,long num){
    dim3 threads(1024,1,1);
    dim3 blocks((num+1024-1)/1024,1,1);   
    floattohalf<<<blocks,threads>>>(A,B,num);
}

int main(){
    hipblasHandle_t cublasH = NULL;
    const int m = 2;
    const int n = 4;
    int calculateTime = 100;
    bool calculateError = true;
    if(calculateError)
        calculateTime=1;
    printf("************************m************************\n %d \n*****************************************\n",m);
    dt *h_A = NULL, *d_A = NULL, *d_AAT=NULL;
    half *d_Ah=NULL,*d_Ah2=NULL;
    hipHostAlloc((void**)&h_A,sizeof(dt)*m*n,0);
    for(int i = 0; i < n*m; i++){
        h_A[i] = (dt) rand()*1.0 / RAND_MAX*1.0*2.0 - 1.0;
    }

    cudaStat1 = hipMalloc((void**)&d_A, sizeof(dt)*m*n);
    cudaStat2 = hipMalloc((void**)&d_Ah, sizeof(half)*m*n);
    cudaStat2 = hipMalloc((void**)&d_Ah2, sizeof(half)*m*n);
    cudaStat3 = hipMalloc((void**)&d_AAT, sizeof(dt)*m*m);
    assert(cudaStat1 == hipSuccess);
    assert(cudaStat2 == hipSuccess);
    assert(cudaStat3 == hipSuccess);

    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    cublas_status = hipblasSetMathMode(cublasH, mathMode);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    cudaStat1 = hipMemcpyAsync(d_A, h_A, sizeof(dt)*m*n, hipMemcpyHostToDevice);
    assert(cudaStat1 == hipSuccess);

    // printMatrix_Device(m,n,d_A,m,"A");   
    warmupcu();

    GPUTimer timer0;
    timer0.start();
    // cublas_status = hipblasSgeam(cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, 
    //                             n, m, 
    //                             &alpha, d_A, m,
    //                             &beta, d_A, n,
    //                             d_AT, n);
    // assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    float time0 = timer0.seconds();
    printf("*************Transpose time ****************\n %f \n*******************************\n",time0);
    
    for(int i=0;i<calculateTime;i++){
        cublas_status = hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T,
                               m, m, n,
                               &alpha, d_A, HIP_R_32F, m,
                               d_A, HIP_R_32F, m,
                               &beta, d_AAT, HIP_R_32F, m,
                               HIP_R_32F, algo);
        assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    }
    
    
    float time1 = timer0.seconds();
    float time_cal1 = (timer0.seconds()-time0)/(dt)calculateTime + time0;
    printf("*************Time1 calculate1 ****************\n %f \n*******************************\n",time_cal1);
    // printMatrix_Device(m, m, d_AAT, m, "AAT1");
    float norm_1 = -1;
    if(calculateError){
        norm_1 = 0;
        norm2HH_device(d_AAT, m*m, &norm_1);
    }
    float time2 = timer0.seconds();
    float time_norm = time2-time1;
    // printf("*************norm time1****************\n %f \n*******************************\n",time_norm);

    f2h(d_A,d_Ah,m*n);
    f2h(d_A,d_Ah2,m*n);
    float time3 = timer0.seconds();
    float time_reduce = time3-time2;
    printf("*************Reduce accuracy time****************\n %f \n*******************************\n",time_reduce);
    printMatrix_Device(m,n,d_A,m,"A");
    printMatrix_Device(m, m, d_AAT, m, "AAT");
    // cublas_status = hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N,
    //                        m, m, n,
    //                        &alpha, d_A, HIP_R_32F, m,
    //                        d_AT, HIP_R_32F, n,
    //                        &beta2, d_AAT, HIP_R_32F, m,
    //                        HIP_R_32F, algo);
    // assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    for(int i=0;i<calculateTime;i++){
        cublas_status = hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T,
                           m, m, n,
                           &alpha, d_Ah, HIP_R_16F, m,
                           d_Ah, HIP_R_16F, m,
                           &beta2, d_AAT, HIP_R_32F, m,
                           HIP_R_32F, algo);
        assert(cublas_status == HIPBLAS_STATUS_SUCCESS);
    }
    float time4 = (timer0.seconds()-time3)/(dt)calculateTime+time_reduce;
    printMatrix_Device(m, m, d_AAT, m, "AAT");
    printf("*************time2 calculate2****************\n %f \n*******************************\n",time4);
    
    
    float norm_2 = 0;
    if(calculateError){
        printf("norm_1：  %f\n", norm_1);
        norm2HH_device(d_AAT, m*m, &norm_2);
        printf("norm_2：  %f\n", norm_2);
        printf("*************error****************\n %f \n*******************************\n", norm_2/norm_1);
    }

    if(cublasH ) hipblasDestroy(cublasH);
    if(h_A     ) hipHostFree(h_A); h_A = NULL;
    if(d_A     ) hipFree(d_A); d_A = NULL;
    if(d_Ah   ) hipFree(d_Ah); d_Ah = NULL;
    if(d_Ah2   ) hipFree(d_Ah2); d_Ah2 = NULL;
    if(d_AAT   ) hipFree(d_AAT); d_AAT = NULL;
    // hipDeviceReset();
    return 0;
}