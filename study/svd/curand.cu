#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

__global__ void setup_kernel(hiprandState *state)
{
    int id = threadIdx.x + blockIdx.x * 64;
    /* Each thread gets same seed, a different sequence 
       number, no offset */
    hiprand_init(1234, id, 0, &state[id]);
}

__global__ void generate_normal_kernel(hiprandState *state,
                                int n, 
                                float *result)
{
    int id = threadIdx.x + blockIdx.x * 64;
    float count = 0;
    float2 x;
    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];
    /* Generate pseudo-random normals */
    for(int i = 0; i < n/2; i++) {
        x = hiprand_normal2(&localState);
        /* Check if within one standard deviaton */
        if((x.x > -1.0) && (x.x < 1.0)) {
            count++;
        }
        if((x.y > -1.0) && (x.y < 1.0)) {
            count++;
        }
    }
    /* Copy state back to global memory */
    state[id] = localState;
    /* Store results */
    result[id] += count;
}

int main(int argc, char *argv[])
{

    int i;
    float total;
    hiprandState *devStates;
    float *devResults, *hostResults;
    int sampleCount = 10000;

    /* Allocate space for results on host */
    hostResults = (float *)calloc(64 * 64, sizeof(float));

    /* Allocate space for results on device */
    CUDA_CALL(hipMalloc((void **)&devResults, 64 * 64 * 
              sizeof(float)));

    /* Set results to 0 */
    CUDA_CALL(hipMemset(devResults, 0, 64 * 64 * 
              sizeof(float)));

    /* Allocate space for prng states on device */
    CUDA_CALL(hipMalloc((void **)&devStates, 64 * 64 * 
                  sizeof(hiprandState)));
    
    /* Setup prng states */
    setup_kernel<<<64, 64>>>(devStates);

    /* Generate and use normal pseudo-random  */
    for(i = 0; i < 50; i++) {
            generate_normal_kernel<<<64, 64>>>(devStates, sampleCount, devResults);
    }

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(hostResults, devResults, 64 * 64 * 
        sizeof(float), hipMemcpyDeviceToHost));

    /* Show result */
    total = 0;
    for(i = 0; i < 64 * 64; i++) {
        total += hostResults[i];
        std::cout << hostResults[i] << " ";
    }
    printf("\nFraction of normals within 1 standard deviation was %10.13f\n", 
        (float)total / (64.0f * 64.0f * sampleCount * 50.0f));

    /* Cleanup */
    CUDA_CALL(hipFree(devStates));  
    CUDA_CALL(hipFree(devResults));
    free(hostResults);
    printf("^^^^ kernel_example PASSED\n");
    return EXIT_SUCCESS;
}

    
    // /* Set results to 0 */
    // CUDA_CALL(cudaMemset(devResults, 0, 64 * 64 * 
    //           sizeof(float)));